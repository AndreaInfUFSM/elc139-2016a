#include "hip/hip_runtime.h"
#include <list>
#include <iostream>
#include <limits>
#include <cmath>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/iterator/constant_iterator.h>

#define nThreads 32
#define nBlocks  16

using namespace std;

numeric_limits<double> real;
double delta = sqrt(real.epsilon()), infinity = real.infinity();

struct Vec {
  double x, y, z;
  __host__ __device__ Vec(double x2, double y2, double z2) : x(x2), y(y2), z(z2) {}
};

__host__ __device__ Vec operator+(const Vec &a, const Vec &b){ return Vec(a.x+b.x, a.y+b.y, a.z+b.z); }
__host__ __device__ Vec operator-(const Vec &a, const Vec &b){ return Vec(a.x-b.x, a.y-b.y, a.z-b.z); }
__host__ __device__ Vec operator*(double a, const Vec &b) { return Vec(a*b.x, a*b.y, a*b.z); }
__host__ __device__ double dot(const Vec &a, const Vec &b) { return a.x*b.x + a.y*b.y + a.z*b.z; }
__host__ __device__ Vec unitise(const Vec &a) { return (1 / sqrt(dot(a, a))) * a; }

typedef thrust::tuple<double, Vec> Hit;

struct Ray {
  Vec orig, dir;
 __host__ __device__ Ray(const Vec &o, const Vec &d) : orig(o), dir(d) {}
};

struct Scene {
  virtual ~Scene() {};
  __host__ __device__ virtual Hit intersect(const Hit &, const Ray &) const = 0;
};

struct Sphere : public Scene {
  Vec center;
  double radius;

   Sphere(Vec c, double r) : center(c), radius(r) {}
  ~Sphere() {}

  __device__ double ray_sphere(const Ray &ray) const {
    Vec v = center - ray.orig;
    double b = dot(v, ray.dir), disc = b*b - dot(v, v) + radius * radius;
    if (disc < 0) return infinity;
    double d = sqrt(disc), t2 = b + d;
    if (t2 < 0) return infinity;
    double t1 = b - d;
    return (t1 > 0 ? t1 : t2);
  }

   __host__ __device__ Hit intersect(const Hit &hit, const Ray &ray) const {
    double lambda = ray_sphere(ray);
    if (lambda >= thrust::get<0>(hit)) return hit;
    return Hit(lambda, unitise(ray.orig + lambda*ray.dir - center));
  }
};

typedef thrust::device_vector<Scene *> Scenes;
struct Group : public Scene {
  Sphere bound;
  Scenes child;

  Group(Sphere b, Scenes c) : bound(b), child(c) {}
  ~Group() {
    for (Scenes::const_iterator it=child.begin(); it!=child.end(); ++it)
      delete *it;
  }

 __host__ __device__ Hit intersect(const Hit &hit, const Ray &ray) const {
    Hit hit2=hit;
    double l = bound.ray_sphere(ray);
    if (l >= thrust::get<0>(hit)) return hit;
    for (int i = 0; i < child.size(); i++){
			Scene *tmp = (Scene*) child[i];
			hit2 = tmp->intersect(hit2, ray);
		}
    return hit2;
  }
};

__device__ Hit intersect(const Ray &ray, const Scene &s)
{ return s.intersect(Hit(infinity, Vec(0, 0, 0)), ray); }

__device__ double ray_trace(const Vec &light, const Ray &ray, const Scene &s) {
  Hit hit = intersect(ray, s);
  if (thrust::get<0>(hit) == infinity) return 0;
  double g = dot(thrust::get<1>(hit), light);
  if (g >= 0) return 0.;
  Vec p = ray.orig + thrust::get<0>(hit)*ray.dir + delta*thrust::get<1>(hit);
  Hit tmp = intersect(Ray(p, -1. * light), s);
	return (thrust::get<0>(tmp)  < infinity ? 0 : -g);
}

Scene *create(int level, const Vec &c, double r) {
  Scene *s = new Sphere(c, r);
  if (level == 1) return s;
  Scenes child;
  child.push_back(s);
  double rn = 3*r/sqrt(12.);
  for (int dz=-1; dz<=1; dz+=2)
    for (int dx=-1; dx<=1; dx+=2)
      child.push_back(create(level-1, c + rn*Vec(dx, 1, dz), r/2));
  return new Group(Sphere(c, 3*r), child);
}

long wtime(){
  struct timeval t;
  gettimeofday(&t, NULL);
  
  return t.tv_sec*1000000 + t.tv_usec;
}


// Kernel pra 512 threads
__global__ void kernel_ray(char *buffer, Scene *scene){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int n = 512;
  int ss = 4;
  Vec light = unitise(Vec(-1, -3, 2));
	
	if(tid < n){
    for(int x = 0; x < n; ++x){
      double g = 0.0;
      for(int dx = 0; dx < ss; ++dx){
        for(int dy = 0; dy < ss; ++dy){
					Vec dir(unitise(Vec(x+dx*1./ss-n/2., tid+dy*1./ss-n/2., n)));
					// g += ray_trace(light, Ray(Vec(0, 0, -4), dir), *s);
        }
      }
      buffer[tid*n+x] = char(int( 0.5 + 255.0 * g / (ss * ss)));	
    }  	
  }
}

int main(int argc, char *argv[]) {
  int level = 5;
	int n = 512;

  if (argc == 2) level = atoi(argv[1]);

	Scene *s(create(level, Vec(0, -1, 0), 1));
  Scene *s_gpu = NULL;
  char *buffer_gpu = NULL;
  char *buffer_cpu = (char*) malloc(n*n*sizeof(char));

  // Aloca espaco na memoria da GPU
  hipMalloc((void **) &buffer_gpu, n*n*sizeof(char)); 
	hipMalloc((void **) &s_gpu, sizeof(Scene));

	// Transfere os dados Host -> GPU
	hipMemcpy(s_gpu, s, sizeof(Scene), hipMemcpyHostToDevice);

	// Chama o Kernel da GPU
	kernel_ray <<<nThreads, nBlocks>>> (buffer_gpu, s_gpu);	
	
	// Copia os dados da GPU para o host
	hipMemcpy(buffer_cpu, buffer_gpu, n*n*sizeof(char), hipMemcpyDeviceToHost);
  
	// Escreve o resultado no arquivo
  cout << "P5\n" << n << " " << n << "\n255\n";
  for(int i = 0; i < n*n; i++){
    cout << buffer_cpu[i];
  }
  
  // Desaloca a memoria em GPU
	hipFree(buffer_gpu);
	hipFree(s_gpu);

  delete s;
  return 0;
}
