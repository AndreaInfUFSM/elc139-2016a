#include <list>
#include <iostream>
#include <limits>
#include <cmath>
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define nThreads 32
#define nBlocks  16

using namespace std;

numeric_limits<double> real;
double delta = sqrt(real.epsilon()), infinity = real.infinity();

struct Vec {
  double x, y, z;
  Vec(double x2, double y2, double z2) : x(x2), y(y2), z(z2) {}
};
Vec operator+(const Vec &a, const Vec &b){ return Vec(a.x+b.x, a.y+b.y, a.z+b.z); }
Vec operator-(const Vec &a, const Vec &b){ return Vec(a.x-b.x, a.y-b.y, a.z-b.z); }
Vec operator*(double a, const Vec &b) { return Vec(a*b.x, a*b.y, a*b.z); }
double dot(const Vec &a, const Vec &b) { return a.x*b.x + a.y*b.y + a.z*b.z; }
Vec unitise(const Vec &a) { return (1 / sqrt(dot(a, a))) * a; }

typedef pair<double, Vec> Hit;

struct Ray {
  Vec orig, dir;
  Ray(const Vec &o, const Vec &d) : orig(o), dir(d) {}
};

struct Scene {
  virtual ~Scene() {};
  virtual Hit intersect(const Hit &, const Ray &) const = 0;
};

struct Sphere : public Scene {
  Vec center;
  double radius;

   Sphere(Vec c, double r) : center(c), radius(r) {}
  ~Sphere() {}

  double ray_sphere(const Ray &ray) const {
    Vec v = center - ray.orig;
    double b = dot(v, ray.dir), disc = b*b - dot(v, v) + radius * radius;
    if (disc < 0) return infinity;
    double d = sqrt(disc), t2 = b + d;
    if (t2 < 0) return infinity;
    double t1 = b - d;
    return (t1 > 0 ? t1 : t2);
  }

  Hit intersect(const Hit &hit, const Ray &ray) const {
    double lambda = ray_sphere(ray);
    if (lambda >= hit.first) return hit;
    return Hit(lambda, unitise(ray.orig + lambda*ray.dir - center));
  }
};

typedef list<Scene *> Scenes;
struct Group : public Scene {
  Sphere bound;
  Scenes child;

  Group(Sphere b, Scenes c) : bound(b), child(c) {}
  ~Group() {
    for (Scenes::const_iterator it=child.begin(); it!=child.end(); ++it)
      delete *it;
  }

  Hit intersect(const Hit &hit, const Ray &ray) const {
    Hit hit2=hit;
    double l = bound.ray_sphere(ray);
    if (l >= hit.first) return hit;
    for (Scenes::const_iterator it=child.begin(); it!=child.end(); ++it)
      hit2 = (*it)->intersect(hit2, ray);
    return hit2;
  }
};

Hit intersect(const Ray &ray, const Scene &s)
{ return s.intersect(Hit(infinity, Vec(0, 0, 0)), ray); }

double ray_trace(const Vec &light, const Ray &ray, const Scene &s) {
  Hit hit = intersect(ray, s);
  if (hit.first == infinity) return 0;
  double g = dot(hit.second, light);
  if (g >= 0) return 0.;
  Vec p = ray.orig + hit.first*ray.dir + delta*hit.second;
  return (intersect(Ray(p, -1. * light), s).first < infinity ? 0 : -g);
}

Scene *create(int level, const Vec &c, double r) {
  Scene *s = new Sphere(c, r);
  if (level == 1) return s;
  Scenes child;
  child.push_back(s);
  double rn = 3*r/sqrt(12.);
  for (int dz=-1; dz<=1; dz+=2)
    for (int dx=-1; dx<=1; dx+=2)
      child.push_back(create(level-1, c + rn*Vec(dx, 1, dz), r/2));
  return new Group(Sphere(c, 3*r), child);
}

long wtime(){
  struct timeval t;
  gettimeofday(&t, NULL);
  
  return t.tv_sec*1000000 + t.tv_usec;
}


// Kernel pra 512 threads
__global__ void kernel_ray(char *buffer, Scene *scene, Vec *light){
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int n = 512;
  int ss = 4;

	if(tid < n){
    for(int x = 0; x < n; ++x){
      double g = 0.0;
		  for(int dx = 0; dx < ss; ++dx){
	  		for(int dy = 0; dy < ss; ++dy){
					// Vec dir(unitise(Vec(x+dx*1./ss-n/2., tid+dy*1./ss-n/2., n)));
					// g += ray_trace(light, Ray(Vec(0, 0, -4), dir), *s);
	  		}
			}
			buffer[tid*n+x] = char(int( 0.5 + 255.0 * g / (ss * ss)));	
    }  	
  }
}

int main(int argc, char *argv[]) {
  int level = 5;
	int n = 512;

  if (argc == 2) level = atoi(argv[1]);

	Vec light = unitise(Vec(-1, -3, 2));
  Vec *light_gpu = NULL;
	Scene *s(create(level, Vec(0, -1, 0), 1));
  Scene *s_gpu = NULL;
  char *buffer_gpu = NULL;
  char *buffer_cpu = (char*) malloc(n*n*sizeof(char));

  // Aloca espaco na memoria da GPU
  hipMalloc((void **) &buffer_gpu, n*n*sizeof(char)); 
	hipMalloc((void **) &s_gpu, sizeof(Scene));
	hipMalloc((void **) &light_gpu, sizeof(Vec));

	// Transfere os dados Host -> GPU
	hipMemcpy(s_gpu, s, sizeof(Scene), hipMemcpyHostToDevice);
	hipMemcpy(light_gpu, &light, sizeof(Vec), hipMemcpyHostToDevice);

	// Chama o Kernel da GPU
	kernel_ray <<<nThreads, nBlocks>>> (buffer_gpu, s_gpu, light_gpu);	
	
	// Copia os dados da GPU para o host
	hipMemcpy(buffer_cpu, buffer_gpu, n*n*sizeof(char), hipMemcpyDeviceToHost);

  //ini = wtime();
  //for (int y=n-1; y>=0; --y)
    //for (int x=0; x<n; ++x) {
      //double g=0;
      //for (int dx=0; dx<ss; ++dx)
        //for (int dy=0; dy<ss; ++dy) {
          //Vec dir(unitise(Vec(x+dx*1./ss-n/2., y+dy*1./ss-n/2., n)));
          //g += ray_trace(light, Ray(Vec(0, 0, -4), dir), *s);
        //}
      //buffer[p] = char(int(.5 + 255. * g / (ss*ss)));
      //p++;
    //}
  //cout << wtime() - ini << endl;

	// Escreve o resultado no arquivo
  //cout << "P5\n" << n << " " << n << "\n255\n";
  //for(int i = 0; i < n*n; i++){
    //cout << buffer_cpu[i];
  //}
  //cout << wtime() - ini << endl;

  //delete s;
  return 0;
}
